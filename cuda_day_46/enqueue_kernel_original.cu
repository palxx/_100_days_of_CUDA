#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ bool cond(unsigned int val) {
    return (val % 2 == 0);
}

__global__ void enqueue_kernel(unsigned int* input, unsigned int* queue, unsigned int N, unsigned int* queueSize){
unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i< N){
  unsigned int val = input[i];
  if(cond(val)){
    unsigned int j = atomicAdd(queueSize, 1);
    queue[j] = val;
  }
}
}

extern "C" unsigned int enqueue_gpu(unsigned int* input, unsigned int* queue, unsigned int N){
unsigned int* input_d;
unsigned int* queue_d;
unsigned int *queueSize_d;

hipMalloc((void**)&input_d, N*sizeof(unsigned int));
hipMalloc((void**)&queue_d, N*sizeof(unsigned int));
hipMalloc((void**)&queueSize_d, sizeof(unsigned int));
hipDeviceSynchronize();

hipMemcpy(input_d, input, N*sizeof(unsigned int), hipMemcpyHostToDevice);
hipMemcpy(queue_d, queue, N*sizeof(unsigned int), hipMemcpyHostToDevice);
hipMemset(queueSize_d, 0, sizeof(unsigned int));

int numThreadsPerBlock = 256;
int numBlocks = (N + numThreadsPerBlock -1)/numThreadsPerBlock;
enqueue_kernel<<<numBlocks, numThreadsPerBlock>>>(input_d, queue_d, N, queueSize_d);

unsigned int queueSize;
hipMemcpy(&queueSize, queueSize_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

hipMemcpy(queue, queue_d, queueSize * sizeof(unsigned int), hipMemcpyHostToDevice);

hipFree(input_d);
hipFree(queue_d);
hipFree(queueSize_d);

return queueSize;
}
