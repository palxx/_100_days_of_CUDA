#include "hip/hip_runtime.h"
#include "csrgraph.h"
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>

__global__ void bfs_kernel(CSRgraph csrgraph, unsigned int* level, unsigned int* prevFrontier, unsigned int* currFrontier, unsigned int numPrevFrontier,
unsigned int numCurrFrontier,unsigned int currLevel){
  unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
  if(idx<numPrevFrontier){
    unsigned int vertex = prevFrontier[idx];
    for(unsigned int edge = csrgraph.scrPointers[vertex]; edge < csrgraph.scrPointers[vertex+1; edge++]){
      unsigned int neighbor = csrgraph.dst[edge];
      if(atomicCAS(&level[neighbor], UINT_MAX, currLevel)==UINT_MAX){
        unsigned int currFrontierIdx = atomicAdd(numCurrFrontier, 1);
        currFrontier[currFrontierIdx] = neighbor;
      }
    }
  }
}

void bfs_gpu(CSRgraph csrgraph, unsigned int srcVertex, unsigned int* level){
  CSRgraph csrgraph_d;
  csrgraph_d.numVertices = csrgraph.numVertices;
  csrgraph_d.numEdges = csrgraph.numEdges;
  hipMalloc((void**)&csrgraph_d.scrPointers, (csrgraph_d.numEdges+1)*sizeof(unsigned int));
  hipMalloc((void**)&csrgraph_d.dst,csrgraph_d.numEdges*sizeof(unsigned int));
  unsigned int* level_d;
  hipMalloc((void**)&level_d, csrgraph.numVertices * sizeof(unsigned int));
  unsigned int* buffer_1;
  hipMalloc((void**)&buffer_1, csrgraph.numVertices * sizeof(unsigned int));
  unsigned int* buffer_2;
  hipMalloc((void**)&buffer_2, csrgraph_d.numVertices * sizeof(unsigned int));
  unsigned int* numCurrFrontier_d; 
  hipMalloc((void**)&numCurrFrontier_d, sizeof(unsigned int));
  unsigned int* prevfrontier_d;
  unsigned int* currFrontier_d;
  hipDeviceSynchronize();

  hipMemcpy(csrgraph_d.scrPointers, csrgraph.scrPointers, csrgraph_d.numVertices*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(csrgraph_d.dst, csrgraph.dst, csrgraph_d.numVertices*sizeof(unsigned int), hipMemcpyHostToDevice);
  level_d[srcVertex] = 0;
  hipMemcpy(level_d, level, csrgraph_d.numVertices*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(prevfrontier_d, &srcVertex, sizeof(unsigned int), hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  int numPrevFrontier = 1;
  int numThreadsPerBlock = 256;
  for(unsigned int currLevel =1; numPrevFrontier>0; ++currLevel){
    hipMemset(numCurrFrontier_d, 0, sizeof(unsigned int));
    int numBlocks = (numPrevFrontier+numThreadsPerBlock-1/numThreadsPerBlock);
    bfs_kernel<<<numBlocks, numThreadsPerBlock>>>(csrgraph_d, level_d, prevfrontier_d, currFrontier_d, numPrevFrontier, numCurrFrontier_d);
    unsigned int* tmp = prevfrontier_d;
    prevfrontier_d = currFrontier_d;
    currFrontier_d = tmp;
    hipMemcpy(&numPrevFrontier, numCurrFrontier_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
  }

  hipDeviceSynchronize;
  hipMemcpy(level, level_d, csrgraph_d.numVertices*sizeof(unsigned int), cudamemcpyDeviceToHost);


    // Free device memory.
    hipFree(csrgraph_d.scrPointers);
    hipFree(csrgraph_d.dst);
    hipFree(numCurrFrontier_d);
    hipFree(level_d);
    hipFree(prevFrontier_d);
    hipFree(currFrontier_d);
}
