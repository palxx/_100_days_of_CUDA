#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_DIM 8
#define IN_TD BLOCK_DIM
#define OUT_TD (IN_TD - 2)

__global__ void stencil_kernel(float* in, float* out, unsigned int N) {
    // Compute global indices using threadIdx within each block.
    unsigned int i = blockIdx.z * OUT_TD + threadIdx.z - 1;
    unsigned int j = blockIdx.y * OUT_TD + threadIdx.y - 1;
    unsigned int k = blockIdx.x * OUT_TD + threadIdx.x - 1;
    int C0 = 1;
    int C1 = 2;
    if(i >= 0 && j >=0 && k >= 0){
    __shared__ float in_s[IN_TD][IN_TD][IN_TD] = in[i * N * N + j * N + k];
    }

    
    // Process only interior points.
    if (i >= 1 && i < N - 1 &&
        j >= 1 && j < N - 1 &&
        k >= 1 && k < N - 1) {
        out[i * N * N + j * N + k] = C0 * in[i * N * N + j * N + k] +
          C1 * ( in[i * N * N + j * N + (k + 1)] +
                 in[i * N * N + j * N + (k - 1)] +
                 in[i * N * N + (j - 1) * N + k] +
                 in[i * N * N + (j + 1) * N + k] +
                 in[(i - 1) * N * N + j * N + k] +
                 in[(i + 1) * N * N + j * N + k] );
    }
}

int main() {
    const unsigned int N = 128;
    float *in, *out;
    
    // Allocate host memory.
    in = (float*)malloc(N * N * N * sizeof(float));
    out = (float*)malloc(N * N * N * sizeof(float));
    if (!in || !out) {
        fprintf(stderr, "Host memory allocation failed\n");
        return -1;
    }
    
    // Initialize the input array.
    for (unsigned int i = 0; i < N * N * N; i++) {
        in[i] = 1.0f; // Set to any appropriate value.
    }
    
    float *in_d, *out_d;
    hipEvent_t start, stop;
    float elapsedTime;
    
    // Create CUDA events.
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Allocate device memory and time it.
    hipEventRecord(start, 0);
    hipMalloc((void**)&in_d, N * N * N * sizeof(float));
    hipMalloc((void**)&out_d, N * N * N * sizeof(float));
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Allocate Time: %f ms\n", elapsedTime);
    
    // Copy data to GPU and time it.
    hipEventRecord(start, 0);
    hipMemcpy(in_d, in, N * N * N * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Copy to GPU time: %f ms\n", elapsedTime);
    
    // Launch the kernel and time it.
    dim3 threadPerBlock(BLOCK_DIM, BLOCK_DIM, BLOCK_DIM);
    dim3 gridSize((N + OUT_TD - 1) / OUT_TD,
                  (N + OUT_TD - 1) / OUT_TD,
                  (N + OUT_TD - 1) / OUT_TD);
    
    hipEventRecord(start, 0);
    stencil_kernel<<<gridSize, threadPerBlock>>>(in_d, out_d, N);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel Time: %f ms\n", elapsedTime);
    
    // Copy data from GPU and time it.
    hipEventRecord(start, 0);
    hipMemcpy(out, out_d, N * N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Copy from GPU time: %f ms\n", elapsedTime);
    
    // Cleanup.
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(in_d);
    hipFree(out_d);
    free(in);
    free(out);
    
    return 0;
}
