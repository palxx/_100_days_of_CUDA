#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_DIM 1024
#define thread_coa 4 

__global__ void reduce_kernel(float* input, float* partialSums, unsigned int N){
    unsigned int segment = blockIdx.x * blockDim.x * 2 * thread_coa;
    unsigned int i = segment + threadIdx.x;

    __shared__ float input_s[BLOCK_DIM];
    input_s[threadIdx.x] = input[i] + input[i+BLOCK_DIM];
    float sum = 0.0f;
    for(unsigned int j = 0; j < thread_coa; ++j){
      sum += input[i+j*BLOCK_DIM];
    }
    input_s[threadIdx.x] = sum;
    __syncthreads;

    for(unsigned int stride = BLOCK_DIM/2; stride > 0; stride /= 2){
        if(threadIdx.x % stride == 0){
            input_s[i] += input_s[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0){
        partialSums[blockIdx.x] = input[i];
    }
}

int main(){
    const unsigned int N = 128;
    float *in, *out;

    // Allocate host memory.
    in = (float*)malloc(N * N * N * sizeof(float));
    out = (float*)malloc(N * N * N * sizeof(float));
    if (!in || !out) {
        fprintf(stderr, "Host memory allocation failed\n");
        return -1;
    }

    // Initialize the input array.
    for (unsigned int i = 0; i < N * N * N; i++) {
        in[i] = 1.0f; // Set to any appropriate value.
    }

    float *in_d, *out_d;
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipMalloc((void**)&in_d, N * sizeof(float));
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Allocate Time: %f ms\n", elapsedTime);

    hipEventRecord(start, 0);
    hipMemcpy(in_d, in, N * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Copy to GPU time: %f ms\n", elapsedTime);

    hipEventRecord(start, 0);
    const unsigned int numThreadsBlock = BLOCK_DIM;
    const unsigned int elePerBlock = 2 * numThreadsBlock * thread_coa;
    const unsigned int numBlocks = (N + elePerBlock - 1) / elePerBlock;
    float* partialSums = (float*)malloc(numBlocks * sizeof(float));
    float *partialSums_d;
    hipMalloc((void**)&partialSums_d, numBlocks * sizeof(float));
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Partial sums allocation time: %f ms\n", elapsedTime);

    hipEventRecord(start, 0);
    reduce_kernel<<<numBlocks, numThreadsBlock>>>(in_d, partialSums_d, N);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel Execution Time: %f ms\n", elapsedTime);

    hipEventRecord(start, 0);
    hipMemcpy(partialSums, partialSums_d, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Copy from GPU: %f ms\n", elapsedTime);

    hipEventRecord(start, 0);
    float sum = 0.0f;
    for (unsigned int i = 0; i < numBlocks; i++) {
        sum += partialSums[i];
    }
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Print Partial Sums: %f ms\n", elapsedTime);

    // Cleanup.
    hipEventRecord(start, 0);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(in_d);
    hipFree(partialSums_d);
    free(partialSums);
    free(in);
    free(out);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("freeup end time: %f ms\n", elapsedTime);

    return 0;
}
