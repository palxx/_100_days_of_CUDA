
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_DIM 1024

// ====================== Timer Utilities ======================

typedef struct {
    hipEvent_t start, stop;
} Timer;

void startTime(Timer* t) {
    hipEventCreate(&t->start);
    hipEventCreate(&t->stop);
    hipEventRecord(t->start, 0);
}

void stopTime(Timer* t) {
    hipEventRecord(t->stop, 0);
    hipEventSynchronize(t->stop);
}

void printElapsedTime(Timer t, const char* label) {
    float elapsed;
    hipEventElapsedTime(&elapsed, t.start, t.stop);
    printf("%s: %.4f ms\n", label, elapsed);
    hipEventDestroy(t.start);
    hipEventDestroy(t.stop);
}

// ====================== Scan Kernel ======================

__global__ void scan_kernel(float* input, float* output, float* partialSums, unsigned int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = input[i];
    __syncthreads();

    for (unsigned int stride = 1; stride <= BLOCK_DIM / 2; stride *= 2) {
        float v;
        if (threadIdx.x >= stride) {
            v = output[i - stride];
        }
        __syncthreads();
        if (threadIdx.x >= stride) {
            output[i] += v;
        }
        __syncthreads();
    }

    if (threadIdx.x == BLOCK_DIM - 1) {
        partialSums[blockIdx.x] = output[i];
    }
}

// ====================== Add Kernel ======================

__global__ void add_kernel(float* output, float* partialSums, unsigned int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x > 0) {
        output[i] += partialSums[blockIdx.x - 1];
    }
}

// ====================== Recursive GPU Scan ======================

void scan_gpu_d(float* input_d, float* output_d, unsigned int N) {
    Timer timer;

    const unsigned int numThreadsPerBlock   = BLOCK_DIM;
    const unsigned int numElementsPerBlock  = numThreadsPerBlock;
    const unsigned int numBlocks            = (N + numElementsPerBlock - 1) / numElementsPerBlock;

    // Allocate partial sums
    startTime(&timer);
    float* partialSums_d;
    hipMalloc((void**)&partialSums_d, numBlocks * sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Partial sums allocation time");

    // Scan kernel
    startTime(&timer);
    scan_kernel<<<numBlocks, numThreadsPerBlock>>>(input_d, output_d, partialSums_d, N);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time");

    // Scan partial sums recursively if needed
    if (numBlocks > 1) {
        scan_gpu_d(partialSums_d, partialSums_d, numBlocks);
        add_kernel<<<numBlocks, numThreadsPerBlock>>>(output_d, partialSums_d, N);
        hipDeviceSynchronize();
    }

    hipFree(partialSums_d);
}

// ====================== Main ======================

int main() {
    const unsigned int N = 1 << 20;

    float* input  = (float*)malloc(N * sizeof(float));
    float* output = (float*)malloc(N * sizeof(float));

    for (unsigned int i = 0; i < N; ++i) {
        input[i] = 1.0f;
    }

    float *input_d, *output_d;
    hipMalloc((void**)&input_d, N * sizeof(float));
    hipMalloc((void**)&output_d, N * sizeof(float));
    hipMemcpy(input_d, input, N * sizeof(float), hipMemcpyHostToDevice);

    scan_gpu_d(input_d, output_d, N);
    hipMemcpy(output, output_d, N * sizeof(float), hipMemcpyDeviceToHost);

    // Verification
    bool correct = true;
    for (unsigned int i = 0; i < N; ++i) {
        if (fabs(output[i] - (i + 1)) > 1e-5f) {
            printf("Mismatch at %u: got %f, expected %f\n", i, output[i], (float)(i + 1));
            correct = false;
            break;
        }
    }

    if (correct) {
        printf("Scan verified successfully!\n");
    }

    hipFree(input_d);
    hipFree(output_d);
    free(input);
    free(output);
    return 0;
}
