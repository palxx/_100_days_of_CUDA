#include "hip/hip_runtime.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include "coo_matrix.h"
#define THREAD_PER_BLOCK 1024

// Forward declaration
__global__ void cooKernel(const cooMatrix mat, const float *x, float *y);

void coo_kernel(const cooMatrix &h_cooMat, const float *h_invec, float *outvec) {
  cooMatrix d_cooMat;
  d_cooMat.numRows = h_cooMat.numRows;
  d_cooMat.numCols = h_cooMat.numCols;
  d_cooMat.numNonZeros = h_cooMat.numNonZeros;

  float *d_inVec = nullptr;
  float *d_outVector = nullptr;

  // Allocate device memory using correct sizes
  hipMalloc((void**)&d_cooMat.rowId, d_cooMat.numNonZeros * sizeof(unsigned int));
  hipMalloc((void**)&d_cooMat.colId, d_cooMat.numNonZeros * sizeof(unsigned int));
  hipMalloc((void**)&d_cooMat.values, d_cooMat.numNonZeros * sizeof(float));
  hipMalloc((void**)&d_inVec, d_cooMat.numCols * sizeof(float));
  hipMalloc((void**)&d_outVector, d_cooMat.numRows * sizeof(float));

  // Copy host data to device memory with the correct source and sizes
  hipMemcpy(d_cooMat.rowId, h_cooMat.rowId, d_cooMat.numNonZeros * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_cooMat.colId, h_cooMat.colId, d_cooMat.numNonZeros * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_cooMat.values, h_cooMat.values, d_cooMat.numNonZeros * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_inVec, h_invec, d_cooMat.numCols * sizeof(float), hipMemcpyHostToDevice);

  // Initialize output vector on device to zero
  hipMemset(d_outVector, 0, d_cooMat.numRows * sizeof(float));

  dim3 gridsize((d_cooMat.numNonZeros + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);

  // Launch the kernel
  cooKernel<<<gridsize, THREAD_PER_BLOCK>>>(d_cooMat, d_inVec, d_outVector);

  // Copy result back to host
  hipMemcpy(outvec, d_outVector, d_cooMat.numRows * sizeof(float), hipMemcpyDeviceToHost);

  // Free allocated device memory
  hipFree(d_cooMat.rowId);
  hipFree(d_cooMat.colId);
  hipFree(d_cooMat.values);
  hipFree(d_inVec);
  hipFree(d_outVector);
}
